
#include <hip/hip_runtime.h>
class Schema
{
public:
    int vehicle_id;//the process ID of the car serves as the vehicle ID.
    int database_index;//fixed constant for mapping to the database purposes.
    double oil_life_pct;
    double tire_p_rl;
    double tire_p_rr;
    double tire_p_fl;
    double tire_p_fr;
    double batt_volt;
    double fuel_percentage;
    bool accel;
    bool seatbelt;
    bool hard_brake;
    bool door_lock;
    bool gear_toggle;
    bool clutch;
    bool hard_steer;
    double speed;
    double distance;
    Schema(){}
};
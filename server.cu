#include <string>
#include <sstream>
#include <vector>
#include <map>
#include <iostream>
#include <chrono>
#include <fstream>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <sys/shm.h>
#include <signal.h>
#include <mutex> //to forbid concurrent reads and writes.
#include <thread>//4 threads: one listener,one moderator,one query API and one message resolver. 
#include <fcntl.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>
#include "Expression.tab.cuh" //to parse any input queries.
#include "proj_types.cuh" //types
SelectQuery* process_query(std::string);
void initialize(int,int,int*,std::map<int,int>*);
Table* t;
GPSSystem* gps_object;
int numberOfCars;
int numberOfVertices;
int numberOfRows;
std::map<int,int>* car_map;
// Aux function to print cuda error:
void cudaErr(){
    // Get last error:
    hipError_t err = hipGetLastError();
    printf("error=%d, %s, %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
}
void listener(int* fd)
{
    //listen on fd[0].
    Schema s(10);
    while(read(fd[0],&s,sizeof(Schema)))
    {
        t->update_worklist(s);
    }
}
void display(ExpressionNode* curr_node,ExpressionNode* parent)
{
    if(curr_node == NULL)
        return;
    std::cout<<curr_node<<":\n";
    if(curr_node->exp_operator != NULL)
        std::cout<<"Opcode: "<<curr_node->exp_operator<<'\n';
    if(curr_node->column_name != NULL)
        std::cout<<"Column name: "<<curr_node->column_name<<'\n';
    std::cout<<"Value,Type,Parent "<<curr_node->value<<" "<<curr_node->type_of_expr<<" "<<parent<<'\n';
    display(curr_node->left_hand_term,curr_node);
    display(curr_node->right_hand_term,curr_node);
    
}

void show_normal_query(const std::vector<Schema>& selected_rows,SelectQuery* select_query)
{
    for(Schema s: selected_rows)
    {
        std::cout<<"{";
        for(char* it: *(select_query->select_columns))
        {
            if(str_equal(it,"vehicle_id"))
                std::cout<<"vehicle_id:"<<s.vehicle_id<<",";
            if(str_equal(it,"oil_life_pct"))
                std::cout<<"oil_life_pct:"<<s.oil_life_pct<<",";
            if(str_equal(it,"tire_p_rl"))
                std::cout<<"tire_p_rl:"<<s.tire_p_rl<<",";
            if(str_equal(it,"tire_p_rr"))
                std::cout<<"tire_p_rr:"<<s.tire_p_rr<<",";
            if(str_equal(it,"tire_p_fl"))
                std::cout<<"tire_p_fl:"<<s.tire_p_fl<<",";
            if(str_equal(it,"tire_p_fr"))
                std::cout<<"tire_p_fr:"<<s.tire_p_fr<<",";
            if(str_equal(it,"batt_volt"))
                std::cout<<"batt_volt:"<<s.batt_volt<<",";
            if(str_equal(it,"fuel_percentage"))
                std::cout<<"fuel_percentage:"<<s.fuel_percentage<<",";
            if(str_equal(it,"accel"))
                std::cout<<"accel:"<<s.accel<<",";
            if(str_equal(it,"seatbelt"))
                std::cout<<"seatbelt:"<<s.seatbelt<<",";
            if(str_equal(it,"hard_brake"))
                std::cout<<"hard_brake:"<<s.hard_brake<<",";
            if(str_equal(it,"door_lock"))
                std::cout<<"door_lock:"<<s.door_lock<<",";
            if(str_equal(it,"clutch"))
                std::cout<<"clutch:"<<s.clutch<<",";
            if(str_equal(it,"hard_steer"))
                std::cout<<"hard_steer:"<<s.hard_steer<<",";
            if(str_equal(it,"speed"))
                std::cout<<"speed:"<<s.speed<<",";
            if(str_equal(it,"distance"))
                std::cout<<"distance:"<<s.distance<<",";
            if(str_equal(it,"origin_vertex"))
                std::cout<<"origin_vertex:"<<s.origin_vertex<<",";
            if(str_equal(it,"destination_vertex"))
                std::cout<<"destination_vertex:"<<s.destination_vertex<<",";
        }
        std::cout<<"}\n";
    }
}

void show_aggregate_query(const std::pair<std::vector<std::vector<std::pair<double,double>>>,std::vector<std::string>>& v,SelectQuery* select_query)
{
    //std::cout<<"Hey,got here! "<<v.second.size()<<'\n';
    int i = 0,j=0;
    for(i=0;i<v.first.size();i++)
    {
        std::cout<<v.second[i]<<":\t{";
        int sz = v.first[i].size();
        for(j=0;j<sz-1;j++)
            std::cout<<"("<<v.first[i][j].first<<","<<v.first[i][j].second<<"),";
        std::cout<<"("<<v.first[i][sz-1].first<<","<<v.first[i][sz-1].second<<")}\n";
    }
}

void show(SelectQuery* sq)
{
    if(sq->select_columns != NULL)
        for(auto it: *(sq->select_columns))
            std::cout<<it<<' ';
    if(sq->aggregate_columns != NULL)
        for(auto it: *(sq->aggregate_columns)){
            std::cout<<"("<<it.first<<" "<<it.second<<"),";
            std::cout<<'\n';
        }
    std::cout<<sq->limit_term<<'\n';
    display(sq->select_expression,NULL);
}

void request_resolver(int* file_descriptor)
{
    request_body* rb = new request_body(0,0,0);
    while(read(file_descriptor[0],rb,sizeof(request_body)))
    {
    //fuel routing or garage
        //std::cout<<"Resolving garage request from "<<rb->sending_car<<'\n';
        int fd = shm_open("vertex_type",O_RDONLY,0666);
        ftruncate(fd,numberOfVertices*sizeof(int));//each node has an associated type. 
        int* type_array = (int*)mmap(0,numberOfVertices*sizeof(int),PROT_READ,MAP_SHARED,fd,0);
        int sending_car = rb->sending_car;//this car's pid. 
        std::map<int,int> current_position = t->get_latest_position();
        int curr_pos = current_position[sending_car];
        std::set<int> dropped_vertices;
        for(int i = 0;i < numberOfVertices;i++)
        {
            double y = (double)rand();
            if(y/RAND_MAX < 0.5 && type_array[i] == 1 && i != curr_pos)//adjust manually
                dropped_vertices.insert(i);//have to ensure that this does not have current position of the car itself!
        }
        close(fd);
        //dropped vertices haveto be computed randomly. 
        std::vector<int> path = gps_object->findGarageOrBunk(curr_pos,rb->request_type,dropped_vertices);
        char c[20];
        // std::cout<<"Path for "<<rb->sending_car<<": ";
        // for(auto it: path)
        //     std::cout<<it<<' ';
        // std::cout<<'\n';
        if(path.size() <= 1)
            continue;//don't send.
        sprintf(c,"shm_1_%d",sending_car);
        fd = shm_open(c,O_CREAT | O_RDWR,0666);
        ftruncate(fd,4);
        int* ptr = (int*)mmap(0,sizeof(int),PROT_READ | PROT_WRITE,MAP_SHARED,fd,0);
        *ptr = 1;
        close(fd);
        c[4] = '2';
        fd = shm_open(c,O_CREAT | O_RDWR,0666);
        ftruncate(fd,4);
        ptr = (int*)mmap(0,sizeof(int),PROT_READ | PROT_WRITE,MAP_SHARED,fd,0);
        *ptr = rb->anomaly_flag;
        close(fd);
        c[4] = '3';
        fd = shm_open(c,O_CREAT | O_RDWR, 0666);
        ftruncate(fd,4);
        ptr = (int*)mmap(0,sizeof(int),PROT_READ|PROT_WRITE,MAP_SHARED,fd,0);
        *ptr = path.size();
        close(fd);
        c[4] = '4';
        fd = shm_open(c,O_CREAT|O_RDWR,0666);
        ftruncate(fd,sizeof(int)*path.size());
        ptr = (int*)mmap(0,path.size(),PROT_READ|PROT_WRITE,MAP_SHARED,fd,0);
        for(int i = 0;i < path.size();i++)
            ptr[i] = path[i];
        close(fd);
        kill(sending_car,SIGUSR1);//send the updated path back to the end user.
    }
}

void query_resolver(int* file_descriptor)//pipe to write to request resolver
{
    std::this_thread::sleep_for(std::chrono::seconds(3));
    std::string s;
    std::ifstream inp;
    inp.open("query.txt",std::ifstream::in);
    while(std::getline(inp,s))
    {
        if(s == "KILL")
            break;
        else if(s.find("CONVOY",0) == 0)
        {
            //convoy query. Else, it's a select query.
            std::stringstream inp_q(s);
            inp_q>>s;//remove the word convoy
            int car_cnt;
            inp_q>>car_cnt;
            bool fl = false;
            std::vector<int> v;
            if(car_cnt <= 0)
                fl = true;
            else
            {
                v.resize(car_cnt);
            }
            for(int i = 0;i < car_cnt;i++)
            {
                int y;
                inp_q>>y;
                if(y <= 0 || y > numberOfCars)
                {
                    fl = true;
                    break;
                }
                v[i] = y;
            }
            if(fl)
                continue;//rejected query.
            
            //convoy request. Proceed to read the set of cars in the convoy, and resolve the same.
            std::cout<<"Convoy query!\n";
            std::map<int,int> returned_details = t->get_latest_position();//gets latest position of all cars. Lock needed?
            std::map<int,int> car_details;
            for(int it: v)
            {
                std::cout<<it<<'\n';
                int x = car_map->find(it)->second;
                car_details[x] = returned_details[x];
            }
            gps_object->convoyNodeFinder(car_details);//takes care of what is needed, including sending a signal to all.
        }
        else
        {
            SelectQuery* sq = process_query(s);
            if(sq == NULL){
                std::cout<<"Ill-formatted query!\n";
                continue;
            }
            //show(sq);
            if(sq->aggregate_columns != NULL)
            {
                std::pair<std::vector<std::vector<std::pair<double,double>>>,std::vector<std::string>> v = t->aggregate_select(sq);
                show_aggregate_query(v,sq);
            }
            else
            {
                std::vector<Schema> v = t->normal_select(sq);
                show_normal_query(v,sq);
            }
            //t->PrintDatabase();
        }
        std::this_thread::sleep_for(std::chrono::seconds(2));
    }
    inp.close();
}

int main(int argc, char* argv[])
{
    int max_wl_size;
    std::ifstream input_file;
    input_file.open(argv[1],std::ifstream::in);
    input_file >>numberOfRows>>max_wl_size>>numberOfCars>>numberOfVertices;
    //std::cout<<numberOfRows<<" "<<max_wl_size<<" "<<numberOfCars<<" "<<numberOfVertices<<'\n'; 
    int f[2];
    pipe(f);
    int request_fd[2];
    pipe(request_fd);
    t = new Table(numberOfRows,numberOfCars,max_wl_size,request_fd);
    int fd = shm_open("adjacency_matrix",O_CREAT|O_RDWR,0666);
    ftruncate(fd,numberOfVertices*numberOfVertices*sizeof(int));
    int* hostAdjacencyMatrix = (int*)mmap(0,numberOfVertices*numberOfVertices*sizeof(int),PROT_READ|PROT_WRITE,MAP_SHARED,fd,0);
    for(int i = 0; i < numberOfVertices; i ++){
        for(int j = 0; j < numberOfVertices; j ++){
            input_file >> hostAdjacencyMatrix[i*numberOfVertices+j];
            if(hostAdjacencyMatrix[i*numberOfVertices+j] < 0) hostAdjacencyMatrix[i*numberOfVertices+j] = INT_MAX;
        }
    }
    close(fd);
    car_map = new std::map<int,int>();
    fd = shm_open("vertex_type",O_CREAT|O_RDWR,0666);
    ftruncate(fd,numberOfVertices*sizeof(int));//each node has an associated type. 
    int* type_array = (int*)mmap(0,numberOfVertices*sizeof(int),PROT_READ|PROT_WRITE,MAP_SHARED,fd,0);
    for(int i = 0;i < numberOfVertices;i++)
        input_file >> type_array[i];//1 for normal, 2 for fuel station, 3 for garage
    close(fd);
    gps_object = new GPSSystem(numberOfVertices, hostAdjacencyMatrix);
    input_file.close();
    std::thread t1(initialize,numberOfCars,numberOfVertices,f,car_map);//creates and runs the cars.
    std::thread t2(listener,f);//listens for server messages.
    std::thread t3(query_resolver,request_fd);
    std::thread t4(request_resolver,request_fd);
    t1.join();
    t2.join();
    t3.join();
    t4.join();
    return 0;//end of program
}
